#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>

#define BLOCK_WIDTH 16

#define A2(i,j) A[(i)*K + (j)]   // A is M×K
#define B2(i,j) B[(i)*N + (j)]   // B is K×N
#define C2(i,j) C[(i)*N + (j)]   // C is M×N

// Think of the three dimensions x,y,z of threadId as being column-major, due to the first dimension x being the one that’s continuous in warpspace.
// threadId = threadIdx.x+blockDim.x*(threadIdx.y+blockDim.y)
// Threads within same warp would access different rows (non contiguous) and same column (non contiguous)
// Destroy the column major property of waprs. Make then 1D.

__global__
void coalescing_matmul_k(const int M, const int K, const int N, const float *A, const float *B, float *C)
{

    int tid_y = threadIdx.x / BLOCK_WIDTH;
    int tid_x = threadIdx.x % BLOCK_WIDTH;

    int row = blockIdx.y * BLOCK_WIDTH + tid_y;
    int col = blockIdx.x * BLOCK_WIDTH + tid_x;

    // kind of simulating row orderness of threadidx after destroying its column-orderness.

    if (row < M && col < N) {
        float sum = 0.0f;
        // Dot‐product of A’s row “row” and B’s column “col”
        for (int k = 0; k < K; ++k) {
            sum += A2(row, k) * B2(k, col);
        }
        C2(row, col) = sum;
    }
}

#undef A2
#undef B2
#undef C2

torch::Tensor coalescing_matmul(torch::Tensor A, torch::Tensor B)
{
    TORCH_CHECK(A.device().is_cuda(), "A must be a CUDA tensor");
    TORCH_CHECK(B.device().is_cuda(), "B must be a CUDA tensor");
    TORCH_CHECK(A.is_contiguous(), "A must be contiguous");
    TORCH_CHECK(B.is_contiguous(), "B must be contiguous");

    int64_t M = A.size(0);
    int64_t K = A.size(1);
    int64_t N = B.size(1);

    TORCH_CHECK(K == B.size(0), "A's col dimension must match B's row dimension");

    auto C = torch::zeros({M, N}, A.options());

    dim3 block(BLOCK_WIDTH * BLOCK_WIDTH);

    dim3 grid((N + BLOCK_WIDTH - 1) / BLOCK_WIDTH, (M + BLOCK_WIDTH - 1) / BLOCK_WIDTH);

    coalescing_matmul_k<<<grid, block>>>(
        (int)M, 
        (int)K, 
        (int)N,
        A.data_ptr<float>(), // equiv to ctypes
        B.data_ptr<float>(),
        C.data_ptr<float>()
    );

    hipDeviceSynchronize(); // wait till the completion is done

    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, hipGetErrorString(err));

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("coalescing_matmul", &coalescing_matmul, "CUDA coalescing");
}